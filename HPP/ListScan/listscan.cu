#include <iostream>
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <stdlib.h>
#include <stdio.h>
#include <time.h>


const int BLOCK = 256;


__global__ 
void ScanListK(float *I, float *O, int l)
{
    int b = blockIdx.x; 
    int t = threadIdx.x;
    
    __shared__ float tSum[BLOCK*2];
    int start = 2*blockDim.x*b;
        
    if(start+t < l){
          tSum[t] = I[start+t];
          //if(tSum[t]!=I[start+t])printf("Mismatch at %d\n", start+t);
     }
     else{
      tSum[t] = 0.0;//First half
     }
     if(start+blockDim.x+t < l){
          tSum[t+blockDim.x] = I[start+blockDim.x+t];
          //if(tSum[t+blockDim.x]!=I[start+blockDim.x+t])printf("Mismatch at %d\n", start+t+blockDim.x);
     } 
     else{ 
          tSum[t+blockDim.x] = 0.0;//Second half
     }
    __syncthreads();

    //Reduction
    for(int s = 1; s <= BLOCK; s*=2){
        int i = (t+1)*s*2-1;
        if (i < BLOCK*2) tSum[i] += tSum[i-s];         
        __syncthreads();
        
    }
    //Post-reduction
    
    for(int s = BLOCK/2; s > 0; s /= 2){
        __syncthreads();
        int j = (t+1)*s*2-1; // Same as other index
        if(j+s < 2*BLOCK)tSum[j+s] += tSum[j];
    } 
       
    
    if(start+t < l){
        O[start+t] = tSum[t];       
    } 
    if(t+start+blockDim.x < l) 
    O[t+start+blockDim.x] = tSum[t+blockDim.x];
    
}
__host__
void scanList(float *h_I, float * h_O, int h_l){
    
    float *d_I, *d_O;
    int olen;    
    olen = h_l / (BLOCK*2); //The output length equals twice the total of the length divided by width
    if ((h_l - olen*BLOCK*2) > 0) { 
        olen++;
    }
    printf("%d blocks\n", olen);
    hipMalloc((void **) &d_I, sizeof(float)*h_l);
    hipMalloc((void **) &d_O, sizeof(float)*h_l);
    
    
    hipMemcpy(d_I, h_I, sizeof(float)*h_l, hipMemcpyHostToDevice);
    hipError_t error =  hipGetLastError();   
    if(error!=hipSuccess){
        fprintf(stderr,"ERROR1: %s\n", hipGetErrorString(error) );
        
    }
    
    dim3 dimGrid(olen, 1, 1);
    dim3 dimBlock(BLOCK, 1, 1);
    
    ScanListK<<<dimGrid, dimBlock>>>(d_I, d_O, h_l);
 
    hipDeviceSynchronize();
    error = hipGetLastError();
    if(error!=hipSuccess){
        fprintf(stderr,"ERROR: %s\n", hipGetErrorString(error) );
        
    }
    hipMemcpy(h_O, d_O, sizeof(float)*h_l, hipMemcpyDeviceToHost);
    hipFree(d_I);hipFree(d_O);
    if(olen>1){
        for(int i = 1; i < olen; i++){
            float preSum = h_O[(BLOCK*i*2)-1];
	    for(int j = 0; j < 2*BLOCK; j++){
                int idx = (BLOCK*i*2)+j;
                if(idx < h_l){
                    h_O[idx]+=preSum;
                } 
            }
        }
    }
    
}

void populateArray(float a[], int l){
        srand48(time(NULL));
        float prev = drand48()*100;
        float nxt;
	for(int i = 1; i < l; i++){
        	do{
		    nxt = drand48()*10; 
                }while(nxt==prev);
            a[i] = nxt;
            prev = nxt;
	}
}

float absDif(float a, float b){
      float c = a-b;
      if(c < 0)c*=-1;
      return c;
}


int main(){
    int lengths[5] = {128, 256, 200, 1500, 1100};
    //for(int x=0; x < 5; x++){
    //int ilen = lengths[x];
    int ilen = 1500;
    float * I;
    I = new float [ilen];
    populateArray(I, ilen);
    printf("%d items\n", ilen);
    float gtotal[ilen];
    scanList(I,gtotal,ilen);
    float rtotal = 0.0;
    for(int i = 0; i < ilen; i ++){
            rtotal += I[i];
	    I[i]=rtotal;
     }
     for(int i =0; i < ilen; i++){
         float dif = absDif(I[i], gtotal[i]);
         if(dif > 1.0)printf("Mistatake @%d %f\n", i, dif); 
    }
    delete [] I;
    //}
    return 0;
}

