#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include "matrix.h"

const int SIZE = 4;


#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


   
__global__ 
void matrixMultK(float *A, float *B, float *C, int n, int m, int k)
{
    int bx = blockIdx.x; int by = blockIdx.y;
    int tx = threadIdx.x; int ty = threadIdx.y;

    int Row = by*blockDim.y+ty;
    int Col = bx*blockDim.x+tx;
    __shared__ float s_A[SIZE][SIZE]; 
    __shared__ float s_B[SIZE][SIZE];
    
    float Cvalue = 0.0; 
    for(int t = 0; t < (n-1)/SIZE+1; t ++){
        if(t*SIZE+tx < n && Row < m){
            s_A[ty][tx] = A[Row*n+t*SIZE+tx];
        }
        else{
            s_A[ty][tx] = 0.0;
        }
        if(Col < k && t*SIZE+ty< n){
            s_B[ty][tx] = B[Col+(t*SIZE+ty)*k];
        }
        else{
            s_B[ty][tx] = 0.0;
        }
        
        __syncthreads();
        for(int i = 0; i < SIZE; i++){
             Cvalue += s_A[ty][i]*s_B[i][tx];
        };
        __syncthreads();
    }
    if(Row < k && Col < m){
        C[Row*k+Col] = Cvalue;
    }
}
__host__
void matrixMult(float *h_A, float *h_B, float *h_C, int n, int m, int k){
    
    float *d_A, *d_B, *d_C; 
    hipMalloc((void **) &d_A, sizeof(float)*n*m);
    hipMalloc((void **) &d_B, sizeof(float)*n*k);
    hipMalloc((void **) &d_C, sizeof(float)*k*m);    
    
    hipMemcpy(d_A, h_A, sizeof(float)*n*m, hipMemcpyHostToDevice); 
    hipMemcpy(d_B, h_B, sizeof(float)*n*k, hipMemcpyHostToDevice);
    
    dim3 dimGrid((k-1)/SIZE+1, (m-1)/SIZE+1, 1);
    dim3 dimBlock(SIZE, SIZE, 1);
    
    matrixMultK<<<dimGrid, dimBlock>>>(d_A,d_B,d_C,n,m,k);

    hipMemcpy(h_C, d_C, sizeof(float)*k*m, hipMemcpyDeviceToHost);
    hipFree(d_A);hipFree(d_B);hipFree(d_C);
    
}


int main(){
    srand(time(NULL));
    int Ax, Ay, Bx, By;
    do {
    //Ax = SIZE + (rand() % SIZE)*2;
    //Ay = SIZE + (rand() % SIZE)*2;
    //Bx = SIZE + (rand() % SIZE)*2;
    //By = SIZE + (rand() % SIZE)*2;
    Ax = 128;
    Ay = 100;
    Bx = 56;
    By = 128;
    }while(Ax!=By);
    Matrix A (Ax,Ay);
    Matrix B (Bx,By);
    //float x[] = {1.0, 2.0, 3.0, 4.0, 5.0, 6.0, 7.0, 8.0, 9.0};
    //float y[] = {1,0, 0.0, 0.0, 1.0, 0.0, 0.0, 0.0, 1.0, 0.0};
    //Matrix A (3,3,x);
    //Matrix B (3,3,y);
    
    A.print();
    B.print();
    float c [A.y*B.x];
    matrixMult(A.a, B.a, c, A.x, A.y, B.x);
    Matrix C (B.x, A.y, c);
    C.print();
    return 0;
}

