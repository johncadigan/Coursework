#include <iostream>
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <stdlib.h>
#include <stdio.h>
#include <time.h>


__global__ 
void AsyncvecAddK(int *A, int *B, int *C, int len, int offset)
{
	int i = threadIdx.x+blockDim.x*blockIdx.x+offset;
        if(i<len) C[i] = A[i] - B[i];

}

__global__
void vecAddK(int *A, int *B, int *C, int len)
{
	int i = threadIdx.x+blockDim.x*blockIdx.x;
        if(i<len) C[i] = A[i] - B[i];

}


void populateArray(int a[], int l){
        time(NULL);
        int prev = rand() % 10;
        int nxt;
	for(int i = 1; i < l; i++){
        	do{
		    nxt = rand() % 10; 
                }while(nxt==prev);
            a[i] = nxt;
            prev = nxt;
	}
}

__host__
void svecAdd(){
    
    int const items = 1;
    int const len = 1024*1024;
    int const nStreams = 4;
    ///Device query boilerplate
    int deviceCount = 0;
    
    hipError_t error_id = hipGetDeviceCount(&deviceCount);
    
    if (error_id != hipSuccess)
    {
        printf("hipGetDeviceCount returned %d\n-> %s\n", (int)error_id, hipGetErrorString(error_id));
        printf("Result = FAIL\n");
        exit(EXIT_FAILURE);
    }

    // This function call returns 0 if there are no CUDA capable devices.
    if (deviceCount == 0)
    {
        printf("There are no available device(s) that support CUDA\n");
	return;
    }
    else
    {
        printf("Detected %d CUDA Capable device(s)\n", deviceCount);
    }
    int fastestDevice = 0;
    int fastestSpeed = 0;
    int bx = 0;
    int gx = 0;
    for (int dev = 0; dev < deviceCount; ++dev)
    {
        hipSetDevice(dev);
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);
        int speed = deviceProp.multiProcessorCount;
        if(speed > fastestSpeed){
	    fastestDevice = dev;
            fastestSpeed = speed;
            bx = deviceProp.maxThreadsDim[0];
            gx = deviceProp.maxGridSize[0];
	}
    }
    hipSetDevice(fastestDevice);
    
    int BLOCK = 256; 
    while(BLOCK * gx < len && BLOCK < bx){///While current block size is too small  
        BLOCK *= 2;
    }

    //int A[items][len];
    //int B[items][len];
    
    ///float A[SIZE];float B[SIZE];float C[SIZE];
    //for(int i=0; i < items; i++){
        //int a[len];
	//populateArray(a, len);
        //int b[len];
	//populateArray(b, len);
        //for(int j=0; j < len; j++){
        //    A[i][j] = a[j];
        //    B[i][j] = b[j];
        //}
    //}

    int size = len*sizeof(int);
    hipStream_t stream[nStreams];
    
    int * dA;
    int * hA;
    int * dB;
    int * hB;
    int * dC;
    int * hC;
    

    ///Create streams and allocated memory to accomodate one vector
    for (int i = 0; i < nStreams; ++i){
	hipStreamCreate(&stream[i]);
    }

    hipHostMalloc((void**)&hA, size, hipHostMallocDefault); 
    hipMalloc((void **) &dA, size);
    hipMalloc((void **) &dB, size);
    hipHostMalloc((void**)&hB, size, hipHostMallocDefault);
    hipMalloc((void **) &dC, size);
    hipHostMalloc((void**)&hC, size, hipHostMallocDefault);
    
    float gms = 0.0; //Time for all Asynch GPU
    float sgms = 0.0; //Time for all synch GPU
    float cms = 0.0; //Time for all CPU
    hipEvent_t startEvent, stopEvent;
    hipEventCreate(&startEvent);
    hipEventCreate(&stopEvent);
    int segSize = len/nStreams;
    dim3 DimGrid = (segSize-1)/BLOCK + 1;
    dim3 DimBlock = BLOCK;
    
    for(int h = 0; h < items; h++){
        populateArray(hA, len);
        populateArray(hB, len);
        //int * hA = A[h];
        //int * hB = B[h];
        float ms;
        hipEventRecord(startEvent,0);    
	for(int i = 0; i < nStreams; i++){ //transfer and compute with segment size
            int offset = i * segSize;
	    hipMemcpyAsync(&dA[offset], &hA[offset], segSize*sizeof(int), hipMemcpyHostToDevice, stream[i]);
	    hipMemcpyAsync(&dB[offset], &hB[offset], segSize*sizeof(int), hipMemcpyHostToDevice, stream[i]);	
	    AsyncvecAddK<<<DimGrid, DimBlock, 0 , stream[i%nStreams]>>>(dA,dB,dC,len, offset);
            hipMemcpyAsync(&hC[offset], &dC[offset], segSize*sizeof(int), hipMemcpyDeviceToHost, stream[i]);
            hipStreamSynchronize(stream[i]);   
        }
        hipEventRecord(stopEvent, 0);
        hipEventSynchronize(stopEvent);
        hipEventElapsedTime(&ms, startEvent, stopEvent);
        gms+=ms;
    	ms = 0.0;
        dim3 DimSGrid((len-1)/BLOCK + 1);
        dim3 DimSBlock(BLOCK);
        hipEventRecord(startEvent,0);
	hipMemcpy(dA, hA, size, hipMemcpyHostToDevice);
        hipMemcpy(dB, hB, size, hipMemcpyHostToDevice);
        vecAddK<<<DimSGrid, DimSBlock>>>(dA, dB,dC, len);
	hipMemcpy(hC, dC, size, hipMemcpyDeviceToHost);
        hipEventRecord(stopEvent, 0);
        hipEventSynchronize(stopEvent);
        hipEventElapsedTime(&ms, startEvent, stopEvent);
        sgms+=ms;

        time_t start, end;
        time(&start); 
        for(int j = 0; j < len; j++){//cpu
           hC[j]=hA[j]-hB[j];
	}
        time(&end);
        cms += (float) difftime(end, start)*1000;   
     }
    printf("Async GPU: %f\nGPU: %f\nCPU: %f\n", sgms/ (float) items, gms / (float) items, cms / (float) items);
    
    hipFree(dA);hipFree(dB);hipFree(dC);
    for (int i = 0; i < nStreams; ++i)hipStreamDestroy(stream[i]);
}


int main(){

    svecAdd();
    return 0;
}

