#include <iostream>
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <algorithm>


#define HISTOGRAM_LENGTH 256
#define CBLOCK 512


//@@ insert code here
__global__
	void FtoCKernel(float *IIData, unsigned char *ICData, int Ipixels){
	int pos = blockIdx.x*CBLOCK+threadIdx.x;
	if(pos < Ipixels) ICData[pos] = (unsigned char) 255.0 * IIData[pos];
}

__global__
	void RtoGKernel(unsigned char *IIData, unsigned char *ICGData, int Ipixels){
	int i = blockIdx.x*CBLOCK+threadIdx.x*3;
	if(i < Ipixels*3){
		ICGData[i] = (unsigned char) (0.21*IIData[i] + 0.71*IIData[i+1] + 0.07*IIData[i+2] );  
	}
	
}

__global__
	void HKernel(unsigned char * IIData, unsigned int * IHistogram, int Ipixels){
	int b = blockIdx.x *blockDim.x; 
        int t = threadIdx.x;
	__shared__ int pHistogram[HISTOGRAM_LENGTH];
	if(t < HISTOGRAM_LENGTH)pHistogram[t] = 0;
	__syncthreads();
	int i = b+t;
	int s = blockDim.x* gridDim.x;
	while(i < Ipixels){
		atomicAdd( &(pHistogram[IIData[i]]), 1);
		i+=s;
	}
	__syncthreads();
	if(t < HISTOGRAM_LENGTH)atomicAdd( &(IHistogram[t]), pHistogram[t]);	
}

__global__
	void CDFKernel(unsigned int * IHistogram, float * ICDF, float fIarea){ 
    int t = threadIdx.x;
	if(t < HISTOGRAM_LENGTH) ICDF[t] = 0.0;
	//reductio
	for(int s = 1; s <= HISTOGRAM_LENGTH/2; s*=2){
        int i = (t+1)*s*2-1;
        if (i < HISTOGRAM_LENGTH) ICDF[i] += (float) ICDF[i-s] / fIarea;         
        __syncthreads();
        
    }
	//post-reduction
    for(int s =  HISTOGRAM_LENGTH/4; s > 0; s /= 2){
        __syncthreads();
        int j = (t+1)*s*2-1; // Same as other index
        if(j+s < HISTOGRAM_LENGTH)ICDF[j+s] += (float) ICDF[j] / fIarea;
    } 
}

__global__
void CDFminKernel(float * I, float minv) {
    
    int t = threadIdx.x;
    __shared__ int pMin[HISTOGRAM_LENGTH];
    if(t < HISTOGRAM_LENGTH)pMin[t] = I[t];//First half
    __syncthreads();
    for(int s = blockDim.x/2; s > 0; s/=2){
        __syncthreads();
        if(t < s) pMin[t] = (pMin[t] < pMin[t+s]) ? pMin[t+s] : pMin[t];
    }
    minv = pMin[0];
}

void populateArray(float a[], int l){
        srand48(time(NULL));
        float prev = drand48();
        float nxt;
	for(int i = 1; i < l; i++){
        	do{
		    nxt = drand48(); 
                }while(nxt==prev);
            a[i] = nxt;
            prev = nxt;
	}
}

float absDif(float a, float b){
      float c = a-b;
      if(c < 0)c*=-1;
      return c;
}


int main(){
    //int lengths[5] = {5, 10, 20, 40, 50};
    //for(int x=0; x < 5; x++){
    //int ilen = lengths[x];
    int ilen = 5;
    int imageWidth = ilen;
    int imageHeight = ilen;
    int imageChannels = 3;
    float * hostInputImageData;
    float * hostOutputImageData;
    const char * inputImageFile;
    float * dInputImageData;
    unsigned char * dcharImageData;
    unsigned char * charImageData; // host
    unsigned char * dcharGImageData;
    unsigned char * charGImageData; // host
    unsigned int * dImageHistogram;
    float * dImageCDF;
    float dminCDF;
    float * dOutputImageData;
    printf("Size %dx%d\n", ilen, ilen);
    int imageArea = imageWidth*imageHeight;
    float I[imageArea*imageChannels];
    populateArray(I, imageArea*imageChannels);
    //Cuda malloc
    hipMalloc((void **) &dInputImageData, imageArea * imageChannels * sizeof(float));
    hipMalloc((void **) &dcharImageData, imageArea * imageChannels * sizeof(unsigned char));
//    cudaMalloc((void **) &dImageHistogram, HISTOGRAM_LENGTH * sizeof(unsigned int));
//    cudaMalloc((void **) &dImageCDF, HISTOGRAM_LENGTH * sizeof(float));
//    cudaMalloc((void **) &dminCDF, sizeof(float));
//    cudaMalloc((void **) &dOutputImageData, imageArea * sizeof(float));
    //Cuda memcpy
    hipMemcpy(dInputImageData,
               hostInputImageData,
               imageArea * imageChannels *sizeof(float),
               hipMemcpyHostToDevice);
    //Cuda conv 1
    dim3 dimCGrid = (imageArea*imageChannels-1)/CBLOCK + 1; 
    dim3 dimCBlock = CBLOCK;
    FtoCKernel<<<dimCGrid, dimCBlock>>>(dInputImageData,dcharImageData,imageArea*imageChannels);
    hipDeviceSynchronize();    
    //Cuda conv 2
    //dim3 dimGGrid = (imageArea-1)/CBLOCK + 1;
    //dim3 dimGBlock = CBLOCK;
    //RtoGKernel<<<dimGGrid, dimGBlock>>>(dcharImageData, dcharGImageData, imageArea);
    //cudaDeviceSynchronize(); 
    //Cuda mcpy to compare
    printf("Copying back\n");
    hipError_t err = hipMemcpy(charImageData,dcharImageData, imageArea * imageChannels *sizeof(unsigned char), hipMemcpyDeviceToHost);
    printf("Copied\n");
    printf("Error %s\n", hipGetErrorString(err));

//    for(int i = 0; i < imageArea*imageChannels; i ++){
//        unsigned char dres = charImageData[i];
//        unsigned char cres = (unsigned char) 255 * I[i];
//        if(dres != cres)printf("%c != %c at %d\n",dres, cres, i);
//        else printf("OK at %d\n", i);
//    }
    //cudaMemcpy(charGImageData,
    //           dcharGImageData,
    //           imageArea *sizeof(unsigned char),
    //           cudaMemcpyDeviceToHost);
    //for(int i = 0; i < imageArea; i++){
    //		if(charGImageData[i] != (unsigned char) (0.21*charImageData[i] + 0.71*charImageData[i+1] + 0.07*charImageData[i+2] ))printf("Error at %d", i);
    //}

    
    hipFree(dInputImageData);hipFree(dcharImageData);hipFree(dcharImageData);
    hipFree(dImageCDF); hipFree(dOutputImageData);
    //}
    return 0;
}

