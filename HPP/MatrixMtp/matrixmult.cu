#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include "matrix.h"

int SIZE = 16;

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


   
__global__ 
void matrixMultK(float *A, float *B, float *C, int n, int m, int k)
{
    int Row = blockIdx.y*blockDim.y+threadIdx.y;
    int Col = blockIdx.x*blockDim.x+threadIdx.x;
    if ((Row < m) && (Col < k)){
        float Cvalue = 0.0;
        for(int i = 0; i < n; i++){
             Cvalue += A[Row*n+i]*B[Col+i*k];
        }
        C[Row*k+Col] = Cvalue;
        
    }
    
    
}
__host__
void matrixMult(float *h_A, float *h_B, float *h_C, int n, int m, int k){
    
    float *d_A, *d_B, *d_C; 
    hipMalloc((void **) &d_A, sizeof(float)*n*m);
    hipMalloc((void **) &d_B, sizeof(float)*n*k);
    hipMalloc((void **) &d_C, sizeof(float)*k*m);    
    
    hipMemcpy(d_A, h_A, sizeof(float)*n*m, hipMemcpyHostToDevice); 
    hipMemcpy(d_B, h_B, sizeof(float)*n*k, hipMemcpyHostToDevice);
    
    dim3 dimGrid((k-1)/SIZE+1, (m-1)/SIZE+1, 1);
    dim3 dimBlock(SIZE, SIZE, 1);
    
    matrixMultK<<<dimGrid, dimBlock>>>(d_A,d_B,d_C,n,m,k);

    hipMemcpy(h_C, d_C, sizeof(float)*k*m, hipMemcpyDeviceToHost);
    hipFree(d_A);hipFree(d_B);hipFree(d_C);
    
}


int main(){
    srand(time(NULL));
    int Ax, Ay, Bx, By;
    do {
    Ax = SIZE + rand() % SIZE;
    Ay = SIZE + rand() % SIZE;
    Bx = SIZE + rand() % SIZE;
    By = SIZE + rand() % SIZE;
    }while(Ax!=By);
    Matrix A (Ax,Ay);
    Matrix B (Bx,By);
    A.print();
    B.print();
    float c [A.y*B.x];
    matrixMult(A.a, B.a, c, A.x, A.y, B.x);
    Matrix C (B.x, A.y, c);
    C.print();
    return 0;
}

