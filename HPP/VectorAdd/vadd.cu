#include <iostream>
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <stdlib.h>
#include <time.h>

int SIZE = 2;
__global__ 
void vecAddK(float *A, float *B, float *C, int len)
{
	int i = threadIdx.x+blockDim.x*blockIdx.x;
        if(i<len) C[i] = A[i] + B[i];

}

__host__
void vecAdd(float *h_A, float *h_B, float *h_C, int len){
    int size = len*sizeof(float);
    float *d_A, *d_B, *d_C;

    hipMalloc((void **) &d_A, size);
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    
    hipMalloc((void **) &d_B, size);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
    
    hipMalloc((void **) &d_C, size);
    
    dim3 DimGrid((len-1)/256 +1, 1, 1);
    dim3 DimBlock(256, 1, 1);
    vecAddK<<<DimGrid, DimBlock>>>(d_A, d_B,d_C, len);
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
    hipFree(d_A);hipFree(d_B);hipFree(d_C);

}

void populateArray(float a[]){
	for(int i = 0; i < SIZE; i++){
        	srand48(time(NULL)); 
		a[i] = drand48() * 100;
	}
}


int main(){
    float A[SIZE];float B[SIZE];float C[SIZE];
    populateArray(A);
    populateArray(B);

    int block_size = 16;
    vecAdd(A,B,C,SIZE);
    std::cout << A[0] << " + " << B[0] << "=" << C[0] << std::endl; 


	return 0;
}

