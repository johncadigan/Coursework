#include <iostream>
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <stdlib.h>
#include <stdio.h>
#include <time.h>


const int BLOCK = 256;


__global__ 
void AddListK(float *I, float *O, int l)
{
    int b = blockIdx.x; 
    int t = threadIdx.x;
    __shared__ float pSum[BLOCK*2];
    unsigned int start = 2*blockDim.x*b;
    (start+t < l) ? pSum[t] = I[start+t]: pSum[t] = 0.0;//First half
    (start+blockDim.x+t < l) ? pSum[t+blockDim.x] = I[start+blockDim.x+t] : pSum[t+blockDim.x] = 0.0;//Second half
    __syncthreads();
    for(unsigned int s = blockDim.x; s > 0; s/=2){
        __syncthreads();
        (t < s) ? pSum[t] += pSum[t+s] : pSum[t]+= 0;
    }
    //printf("Sum =%f ", pSum[0]);
     
        O[b] = pSum[0];
    
}
__host__
double addList(float *h_I, int h_l){
    
    float *d_I, *d_O;
    int olen;    
    olen = h_l / (BLOCK<<1); //The output length equals twice the total of the length divided by width
    if (olen % (BLOCK<<1)) { 
        olen++;
    }
    
    float h_O[olen];
 
    hipMalloc((void **) &d_I, sizeof(float)*h_l);
    hipMalloc((void **) &d_O, sizeof(float)*olen);
    
      
    hipMemcpy(d_I, h_I, sizeof(float)*h_l, hipMemcpyHostToDevice); 
    
    
    dim3 dimGrid(olen, 1, 1);
    dim3 dimBlock(BLOCK, 1, 1);
    
    AddListK<<<dimGrid, dimBlock>>>(d_I, d_O, h_l);

    hipMemcpy(h_O, d_O, sizeof(float)*olen, hipMemcpyDeviceToHost);
    hipFree(d_I);hipFree(d_O);
    
    double total = 0.0;
    for(int i = 0; i < olen; i ++){
	total+=h_O[i];
    }
    return total;
}

void populateArray(float a[], int l){
        srand48(time(NULL));
        float prev = drand48()*100;
        float nxt;
	for(int i = 1; i < l; i++){
        	do{
		    nxt = drand48()*100; 
                }while(nxt==prev);
            a[i] = nxt;
            prev = nxt;
	}
}


int main(){
    srand(time(NULL));

    //int ilen = (rand() % 6553) * BLOCK;
    int ilen = 2000000;
    float I[ilen];
    populateArray(I, ilen);
    printf("Input length %d", ilen);
    time_t gstart = time(NULL);
    double gtotal = 0.0;
    for(int i = 0; i < 1000; i ++){
        gtotal = addList(I,ilen);
    }
    time_t gstop = time(NULL);
    time_t start = time(NULL);
    double total = 0.0;
    for(int i = 0; i < 1000; i ++){
        total = 0.0;
        for(int i = 0; i < ilen; i ++){
	    total+=I[i];
        }
    }
    time_t stop = time(NULL);
    printf("Average times\n GPU: %f    CPU: %f", difftime(gstop, gstart), difftime(stop, start));
    printf("TOTAL: %f == %f \n DIF: %f", total, gtotal, total-gtotal);

    return 0;
}

